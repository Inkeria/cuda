#include<stdio.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

template<int BLOCK_DIM>
__global__ void reduce(float *A, int n)
{
    __shared__ float shareMem[BLOCK_DIM];
    float tmp = 0;
    if(threadIdx.x < BLOCK_DIM)
    {
        for(int id = threadIdx.x; id < n;id += BLOCK_DIM)
        {
            tmp = tmp + A[id];
        }
        shareMem[threadIdx.x] = tmp;
        __syncthreads();
        for(int rad = BLOCK_DIM >> 1; rad; rad >>= 1)
        {
            if(threadIdx.x < rad) {
                shareMem[threadIdx.x] = shareMem[threadIdx.x] + shareMem[threadIdx.x + rad]; 
            }
            __syncthreads();
        }
        // if(threadIdx.x == 0) ans = shareMem[threadIdx.x];
    }
}

int main()
{
    float *A, *ans;
    const int n = 102400;
    A = (float*) malloc(n * sizeof(float));
    ans = (float *) malloc(sizeof(float));
    for(int i = 0;i < n; ++i){
        A[i] = (n - i + 1) * 1e-1; 
    }
    float *dA;
    hipMalloc((void **)&dA, n * sizeof(float));
    hipMemcpy(A, dA, n * sizeof(float), hipMemcpyHostToDevice);

    // dim3 grid_dim(1, 1, 1);
    // dim3 block_dim(1024, 1, 1);
    // reduce<1024><<<grid_dim, block_dim>>>(dA, n);

    // cudaMemcpy(dA, ans, sizeof(float), cudaMemcpyDeviceToHost);
    printf("%f\n",dA[0]);
}