#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void printHello()
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello GPU by thread:%d",index);
}
int main()
{
    dim3 grid_dim = {1, 1, 1};
    dim3 block_dim = {4, 1, 1};
    printHello<<<grid_dim, block_dim>>>();
    hipDeviceSynchronize();
    return 0;
}
