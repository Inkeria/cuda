#include<stdio.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

template<int BLOCK_DIM>
__global__ void reduce(float *A, int n)
{
    __shared__ float shareMem[BLOCK_DIM];
    float tmp = 0;
    if(threadIdx.x < BLOCK_DIM)
    {
        printf("now run on GPU tread:%d\n",threadIdx.x);
        __syncthreads();
        for(int id = threadIdx.x; id < n;id += BLOCK_DIM)
        {
            tmp = tmp + A[id];
        }
        shareMem[threadIdx.x] = tmp;
        __syncthreads();
        for(int rad = BLOCK_DIM >> 1; rad; rad >>= 1)
        {
            if(threadIdx.x < rad) {
                shareMem[threadIdx.x] = shareMem[threadIdx.x] + shareMem[threadIdx.x + rad]; 
            }
            __syncthreads();
        }
        A[threadIdx.x] = shareMem[threadIdx.x];
    }
    else {
        printf("now not run on GPU tread:%d\n",threadIdx.x);
        __syncthreads();
    }
}

int main()
{
    float *A;
    const int n = 102400;
    A = (float*) malloc(n * sizeof(float));
    // ans = (float *) malloc(sizeof(float));
    for(int i = 0;i < n; ++i){
        A[i] = (n - i + 1) * 1e-2;
    }
    float *dA;
    hipMalloc((void **)&dA, n * sizeof(float));
    printf("%f\n",A[0]);
    hipMemcpy(A, dA, n * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid_dim(1, 1, 1);
    dim3 block_dim(1024, 1, 1);
    reduce<1024><<<grid_dim, block_dim>>>(dA, n);

    hipMemcpy(dA, A, n * sizeof(float), hipMemcpyDeviceToHost);
    printf("%f\n",A[0]);
}