#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

double get_time()
{
    timeval tp;
    gettimeofday(&tp, NULL);
    return (double) (tp.tv_sec + tp.tv_usec*1e-6);
}

__global__ void addKernel(float *A, float *B, float *C, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n)
    {
        C[index] = A[index] + B[index];
    }
}

int main()
{
    float *hostA, *hostB, *hostC;
    const int n = 102400;
    const int N = n * sizeof(float);
    hostA = (float *) malloc(N);
    hostB = (float *) malloc(N);
    hostC = (float *) malloc(N);
    memset(hostA, 0x3f, N);
    memset(hostB, 0x3f, N);
    double st = get_time();
    for(int i = 0;i < n;++i){
        hostC[i] = hostA[i] + hostB[i] + hostC[i];
    }
    double cpu_time = get_time() - st;
    st = get_time();
    float *deviceA, *deviceB, *deviceC;
    hipMalloc((void **)&deviceA, N);
    hipMalloc((void **)&deviceB, N);
    hipMalloc((void **)&deviceC, N);

    hipMemcpy(deviceA, hostA, N, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, N, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float kernel_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int BLOCK_DIM = 1024;
    int num_block_x = n / BLOCK_DIM;
    int num_block_y = 1;

    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);

    addKernel<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&kernel_time, start, stop);
    hipMemcpy(hostC, deviceC, N, hipMemcpyDeviceToHost);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    double gpu_time = get_time() - st;
    printf("kernel_time:%.6f \n", kernel_time);
    printf("cpu_time:%.6f gpu_time:%.6f",cpu_time ,gpu_time);
    return 0;
}