#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>


__global__ void printHello()
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello GPU by thread:%d\n", index);
}
int main()
{
    dim3 grid_dim = {1, 1, 1};
    dim3 block_dim = {4, 1, 1};
    printHello<<<grid_dim, block_dim>>>();
    hipDeviceSynchronize();
    return 0;
}
