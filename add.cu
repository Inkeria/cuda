#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

double get_time()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

__global__ void addKernel(float *A, float *B, float *C, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n)
    {
        C[index] = A[index] + B[index];
    }
}
void initCpu(float *hostA, float *hostB, int n)
{
    for (int i = 0; i < n; i++)
    {
        hostA[i] = 1;
        hostB[i] = 1;
    }
}
int main()
{
    float *hostA, *hostB, *hostC, *ddvvC;
    int n = 102400;
    // const int N = n * sizeof(float);
    hostA = (float *) malloc(n * sizeof(float));
    hostB = (float *) malloc(n * sizeof(float));
    hostC = (float *) malloc(n * sizeof(float));
    ddvvC = (float *) malloc(n * sizeof(float));
    initCpu(hostA, hostB, n);
    // memset(hostA, 0x3f, N);
    // memset(hostB, 0x3f, N);
    double st = get_time();
    for(int i = 0;i < n;++i){
        hostC[i] = hostA[i] + hostB[i] + hostC[i];
    }
    double cpu_time = get_time() - st;
    st = get_time();
    float *deviceA, *deviceB, *deviceC;
    hipMalloc((void **)&deviceA, n * sizeof(float));
    hipMalloc((void **)&deviceB, n * sizeof(float));
    hipMalloc((void **)&deviceC, n * sizeof(float));

    hipMemcpy(deviceA, hostA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float kernel_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int BLOCK_DIM = 1024;
    int num_block_x = n / BLOCK_DIM;
    int num_block_y = 1;

    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);

    addKernel<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&kernel_time, start, stop);
    hipMemcpy(ddvvC, deviceC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    double gpu_time = get_time() - st;
    printf("kernel_time:%.6f \n", kernel_time);
    printf("cpu_time:%.6f \n gpu_time:%.6f\n",cpu_time ,gpu_time);
    
    free(hostA);
    free(hostB);
    free(hostC);
    free(ddvvC);
    return 0;
}