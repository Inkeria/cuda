#include<stdio.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

template<int BLOCK_DIM>
__global__ void reduce(float *A, int n)
{
    __shared__ float shareMem[BLOCK_DIM];
    float tmp = 0;
    // printf("now run on GPU tread:%d\n",threadIdx.x);
    // __syncthreads();
    // if(threadIdx.x < BLOCK_DIM)
    // {
        // printf("now run on GPU tread:%d\n",threadIdx.x);
        // __syncthreads();
        for(int id = threadIdx.x; id < n;id += BLOCK_DIM)
        {
            tmp = tmp + A[id];
        }
        shareMem[threadIdx.x] = tmp;
        __syncthreads();
        for(int rad = BLOCK_DIM >> 1; rad; rad >>= 1)
        {
            if(threadIdx.x < rad) {
                shareMem[threadIdx.x] = shareMem[threadIdx.x] + shareMem[threadIdx.x + rad]; 
            }
            __syncthreads();
        }
        if(blockIdx.x == 0)
        A[threadIdx.x] = shareMem[threadIdx.x];
    // }
    // else {
        // printf("now not run on GPU tread:%d\n",threadIdx.x);
        // __syncthreads();
    // }
}

template<int BLOCK_DIM>
__global__ void shfl_reduce(float *A, int n)
{
    __shared__ float shareMem[BLOCK_DIM];
    float tmp = 0;
    for(int id = threadIdx.x; id < n;id += BLOCK_DIM)
    {
        tmp = tmp + A[id];
    }
    shareMem[threadIdx.x] = tmp;
    __syncthreads();
    __shared__ float val[32];
    tmp = 0;
    tmp += __shfl_down_sync(0xffffffff, tmp, 16);
    tmp += __shfl_down_sync(0xffffffff, tmp, 8);
    tmp += __shfl_down_sync(0xffffffff, tmp, 4);
    tmp += __shfl_down_sync(0xffffffff, tmp, 2);
    tmp += __shfl_down_sync(0xffffffff, tmp, 1);
    if(threadIdx.x >> 5 & 1){
        val[threadIdx.x >> 5] = tmp;
    }
    __syncthreads();
    if(threadIdx.x < 32)
    {
        tmp = val[threadIdx.x];
        tmp += __shfl_down_sync(0xffffffff, tmp, 16);
        tmp += __shfl_down_sync(0xffffffff, tmp, 8);
        tmp += __shfl_down_sync(0xffffffff, tmp, 4);
        tmp += __shfl_down_sync(0xffffffff, tmp, 2);
        tmp += __shfl_down_sync(0xffffffff, tmp, 1);
    }
    __syncthreads();
    if(threadIdx.x == 0){
        A[0] = tmp;
    }
}

int main()
{
    float *A;
    const int n = 102400;
    A = (float*) malloc(n * sizeof(float));
    // ans = (float *) malloc(sizeof(float));
    for(int i = 0;i < n; ++i){
        A[i] = (n - i + 1) * 1e-2;
    }
    float *dA, *ans;
    ans = (float *) malloc(sizeof(float));
    hipMalloc((void **)&dA, n * sizeof(float));
    // printf("%f\n",A[0]);
    hipMemcpy(dA, A, n * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid_dim(100, 1, 1);
    dim3 block_dim(1024, 1, 1);
    shfl_reduce<1024><<<grid_dim, block_dim>>>(dA, n);
    hipDeviceSynchronize();

    hipMemcpy(ans, dA, sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 1;i < n;++i) A[0] += A[i];
    printf("%f\n",*ans);
    printf("%f\n",*A);
}