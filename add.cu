#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

__global__ void addKernel(float *deviceA, float *deviceB, float *deviceC, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n)
    {
        deviceC[index] = deviceA[index] + deviceB[index] + deviceC[index];
    }
}

int main()
{
    float *hostA, *hostB, *hostC;
    const int n = 102400;
    const int N = n * sizeof(float);
    hostA = (float *) malloc(N);
    hostB = (float *) malloc(N);
    hostC = (float *) malloc(N);
    float *deviceA, *deviceB, *deviceC;
    hipMalloc((void **)&deviceA, N);
    hipMalloc((void **)&deviceB, N);
    hipMalloc((void **)&deviceC, N);

    hipMemcpy(deviceA, hostA, N, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, N, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float kernel_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int BLOCK_DIM = 1024;
    int num_block_x = n / BLOCK_DIM;
    int num_block_y = 1;

    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);

    addKernel<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&kernel_time, start, stop);
    hipMemcpy(hostC, deviceC, N, hipMemcpyDeviceToHost);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    printf("kernel_time:%f \n", kernel_time);
    return 0;
}